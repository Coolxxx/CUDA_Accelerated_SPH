#include "hip/hip_runtime.h"
#include "WCSPHSolver.cuh"
#include "handler.h"
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

//#define DEBUG
//#define CONFINE_RANDOM
#define CUDA_MEMCPY_ASYNC
#define CUDA_MEMSET_ASYNC

const int kCudaSortArrayCount = 4;

#if defined(CUDA_MEMCPY_ASYNC) || defined(CUDA_MEMCPY_ASYNC)
const int kCudaMemcpyTime = 7;
#endif // defined(CUDA_MEMCPY_ASYNC) || defined(CUDA_MEMCPY_ASYNC)


////////////////////////////////////////////////////////////////////////////////
// Device array declare
////////////////////////////////////////////////////////////////////////////////
int3 block_offset_host[] = {
	{-1, -1, -1}, {-1, -1, 0}, {-1, -1, 1},
	{-1, 0, -1}, {-1, 0, 0}, {-1, 0, 1},
	{-1, 1, -1}, {-1, 1, 0}, {-1, 1, 1},
	{0, -1, -1}, {0, -1, 0}, {0, -1, 1},
	{0, 0, -1}, {0, 0, 0}, {0, 0, 1},
	{0, 1, -1}, {0, 1, 0}, {0, 1, 0},
	{1, -1, -1}, {1, -1, 0}, {1, -1, 1},
	{1, 0, -1}, {1, 0, 0}, {1, 0, 1},
	{1, 1, -1}, {1, 1, 0}, {1, 1, 1},
};

WCSPHSystem* sph_device = NULL;

int* particle_bid = NULL; // each particle belongs to which block
int* block_pidx = NULL; // first particle index in grid
int* block_pnum = NULL; // particle number in grid
int3* block_offset = NULL;

hiprandState* devStates = NULL;

float3* color = NULL; // color of particles
float3* cur_pos = NULL;
float3* next_pos = NULL;

float* density = NULL;
float* delta_density = NULL;

float* pressure = NULL;
float3* delta_pressure = NULL;

float3* delta_viscosity = NULL;

float3* velocity = NULL;
float3* delta_velocity = NULL;

////////////////////////////////////////////////////////////////////////////////
//
// Init CUDA Device System
//
////////////////////////////////////////////////////////////////////////////////
void InitDeviceSystem(WCSPHSystem* para, float* dens_init, float3* pos_init, float3* velo_init) {

#ifdef DEBUG
	std::cout << "Do InitDeviceSystem" << std::endl;
#endif // DEBUG

	int num = para->particle_num;
#if defined (CUDA_MEMCPY_ASYNC) || defined (CUDA_MEMSET_ASYNC)
	hipStream_t stream[kCudaMemcpyTime];
	int streamnum = 0;
	for (int i = 0; i < kCudaMemcpyTime; i++) {
		checkCudaErrors(hipStreamCreate(&stream[i]));
	}
#endif // CUDA_MEMCPY_ASYNC || CUDA_MEMSET_ASYNC

	checkCudaErrors(hipMalloc(&sph_device, sizeof(WCSPHSystem)));
#ifdef CUDA_MEMCPY_ASYNC
	checkCudaErrors(hipMemcpyAsync(sph_device, para, sizeof(WCSPHSystem), hipMemcpyHostToDevice, stream[streamnum++]));
#else
	checkCudaErrors(hipMemcpy(sph_device, para, sizeof(WCSPHSystem), hipMemcpyHostToDevice));
#endif // CUDA_MEMCPY_ASYNC
	

	checkCudaErrors(hipMalloc(&particle_bid, kCudaSortArrayCount * num * sizeof(int)));

	checkCudaErrors(hipMalloc(&block_pidx, para->block_num * sizeof(int)));
	checkCudaErrors(hipMalloc(&block_pnum, para->block_num * sizeof(int)));

	checkCudaErrors(hipMalloc(&block_offset, 27 * sizeof(int3)));
#ifdef CUDA_MEMCPY_ASYNC
	checkCudaErrors(hipMemcpyAsync(block_offset, block_offset_host, 27 * sizeof(int3), hipMemcpyHostToDevice, stream[streamnum++]));
#else
	checkCudaErrors(hipMemcpy(block_offset, block_offset_host, 27 * sizeof(int3), hipMemcpyHostToDevice));
#endif // CUDA_MEMCPY_ASYNC


	checkCudaErrors(hipMalloc((void**)&devStates, num * sizeof(hiprandState)));

	checkCudaErrors(hipMalloc(&color, num * sizeof(float3)));


	checkCudaErrors(hipMalloc(&cur_pos, num * sizeof(float3)));
#ifdef CUDA_MEMCPY_ASYNC
	checkCudaErrors(hipMemcpyAsync(cur_pos, pos_init, num * sizeof(float3), hipMemcpyHostToDevice, stream[streamnum++]));
#else
	checkCudaErrors(hipMemcpy(cur_pos, pos_init, num * sizeof(float3), hipMemcpyHostToDevice));
#endif // CUDA_MEMCPY_ASYNC


	checkCudaErrors(hipMalloc(&next_pos, num * sizeof(float3)));
#ifdef CUDA_MEMSET_ASYNC
	checkCudaErrors(hipMemsetAsync(next_pos, 0, num * sizeof(float3), stream[3]));
#else
	checkCudaErrors(hipMemset(next_pos, 0, num * sizeof(float3)));
#endif // CUDA_MEMSET_ASYNC


	checkCudaErrors(hipMalloc(&density, num * sizeof(float)));
#ifdef CUDA_MEMCPY_ASYNC
	checkCudaErrors(hipMemcpyAsync(density, dens_init, num * sizeof(float), hipMemcpyHostToDevice, stream[streamnum++]));
#else
	checkCudaErrors(hipMemcpy(density, dens_init, num * sizeof(float), hipMemcpyHostToDevice));
#endif // CUDA_MEMCPY_ASYNC

	checkCudaErrors(hipMalloc(&delta_density, num * sizeof(float)));

	checkCudaErrors(hipMalloc(&pressure, num * sizeof(float)));
#ifdef CUDA_MEMSET_ASYNC
	checkCudaErrors(hipMemsetAsync(pressure, 0, num * sizeof(float), stream[5]));
#else
	checkCudaErrors(hipMemset(pressure, 0, num * sizeof(float)));
#endif // CUDA_MEMSET_ASYNC

	checkCudaErrors(hipMalloc(&delta_pressure, num * sizeof(float3)));

	checkCudaErrors(hipMalloc(&delta_viscosity, num * sizeof(float3)));

	checkCudaErrors(hipMalloc(&velocity, num * sizeof(float3)));
#ifdef CUDA_MEMCPY_ASYNC
	checkCudaErrors(hipMemcpyAsync(velocity, velo_init, num * sizeof(float3), hipMemcpyHostToDevice, stream[streamnum++]));
#else
	checkCudaErrors(hipMemcpy(velocity, velo_init, num * sizeof(float3), hipMemcpyHostToDevice));
#endif // CUDA_MEMCPY_ASYNC
	

	checkCudaErrors(hipMalloc(&delta_velocity, num * sizeof(float3)));

#if defined (CUDA_MEMCPY_ASYNC) || defined (CUDA_MEMSET_ASYNC)
	for (int i = 0; i < kCudaMemcpyTime; i++) {
		checkCudaErrors(hipStreamSynchronize(stream[i]));
		checkCudaErrors(hipStreamDestroy(stream[i]));
	}
#endif // CUDA_MEMCPY_ASYNC || CUDA_MEMSET_ASYNC

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "Finish InitDeviceSystem" << std::endl;
#endif // DEBUG
}

////////////////////////////////////////////////////////////////////////////////
//
// Free CUDA Device System
//
////////////////////////////////////////////////////////////////////////////////
void FreeDeviceSystem(WCSPHSystem* para) {

#ifdef DEBUG
	std::cout << "Do FreeDeviceSystem" << std::endl;
#endif // DEBUG
	delete para;

	checkCudaErrors(hipFree(sph_device));

	checkCudaErrors(hipFree(particle_bid));
	checkCudaErrors(hipFree(block_pidx));
	checkCudaErrors(hipFree(block_pnum));

	checkCudaErrors(hipFree(block_offset));

	checkCudaErrors(hipFree(devStates));

	checkCudaErrors(hipFree(color));

	checkCudaErrors(hipFree(cur_pos));
	checkCudaErrors(hipFree(next_pos));

	checkCudaErrors(hipFree(density));
	checkCudaErrors(hipFree(delta_density));

	checkCudaErrors(hipFree(pressure));
	checkCudaErrors(hipFree(delta_pressure));

	checkCudaErrors(hipFree(delta_viscosity));

	checkCudaErrors(hipFree(velocity));
	checkCudaErrors(hipFree(delta_velocity));

#ifdef DEBUG
	std::cout << "Finish InitDeviceSystem" << std::endl;
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
// CUDA function are implemented here
////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
//
// Compute which block each particle belongs to
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ComputeBid(			WCSPHSystem* para,
									int* particle_bid,
									float3* cur_pos) {

#ifdef DEBUG
	printf("Block #(%d,%d,%d) Do ComputeBid\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	// compute block_id for each particle
	int num = para->particle_num;
	int3 blockDim_i = para->block_dim;
	for (int i = 0; i < num; i++) {
		int3 tmp_bid = make_int3(cur_pos[i] / para->block_size);
		particle_bid[i] = GetBlockIdx1D(tmp_bid, blockDim_i);
	}

#ifdef DEBUG
	printf("Block #(%d,%d,%d) Finish ComputeBid\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
//
// Use Radix sort to place particle in block order
//
////////////////////////////////////////////////////////////////////////////////
__global__ void SortParticles(		WCSPHSystem* para,
									int* particle_bid,
									float* density, float* pressure,
									float3* cur_pos, float3* velocity) {

#ifdef DEBUG
	printf("Block #(%d,%d,%d) Do SortParticles\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	int num = para->particle_num;
	if (blockIdx.x == 0) {
		thrust::stable_sort_by_key(thrust::device, particle_bid + num * 0, particle_bid + num * 1, cur_pos);
	}
	else if (blockIdx.x == 1) {
		thrust::stable_sort_by_key(thrust::device, particle_bid + num * 1, particle_bid + num * 2, density);
	}
	else if (blockIdx.x == 2) {
		thrust::stable_sort_by_key(thrust::device, particle_bid + num * 2, particle_bid + num * 3, pressure);
	}
	else if (blockIdx.x == 3) {
		thrust::stable_sort_by_key(thrust::device, particle_bid + num * 3, particle_bid + num * 4, velocity);
	}
	//else if (blockIdx.x == 4) {
	//	thrust::stable_sort_by_key(thrust::device, particle_bid + num * 4, particle_bid + num * 5, next_pos);
	//}

#ifdef DEBUG
	printf("Block #(%d,%d,%d) Finish SortParticles\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
//
// Compute the index of first particle and the total particle number in each block 
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ComputeBlockIdxPnum(WCSPHSystem* para,
									int* particle_bid, int* block_pidx, int* block_pnum) {

#ifdef DEBUG
	printf("Block #(%d,%d,%d) Do ComputeBlockIdxPnum\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	for (int i = 0; i < para->block_num; i++) {
		block_pidx[i] = -1;
		block_pnum[i] = 0;
	}

	for (int i = 0; i < para->particle_num; i++) {
		if (i == 0 || particle_bid[i] != particle_bid[i - 1]) {
			block_pidx[particle_bid[i]] = i;
		}
		block_pnum[particle_bid[i]]++;
		//if (block_pnum[particle_bid[i]] > para->block_thread_num)
		//	printf("Block %d ERROR, exceed threads number\n", particle_bid[i]);
	}

#ifdef DEBUG
	printf("Block #(%d,%d,%d) Finish ComputeBlockIdxPnum\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
//
// Compute delta value of density, pressure and viscosity for each particle
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ComputeDeltaValue(	WCSPHSystem* para,
									int* block_pidx, int* block_pnum,
									float* delta_density, float* density,  float* pressure,
									int3* block_offset,
									float3* cur_pos, float3* delta_pressure, float3* delta_viscosity, float3* velocity) {

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = para->block_dim;
	int threadIdx_i = threadIdx.x;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i);

	while (threadIdx_i < block_pnum[bid]) {
		// for each particle[i]
		int i = block_pidx[bid] + threadIdx_i;
		// Initialize
		delta_density[i] = 0.0;
		delta_pressure[i] = make_float3(0, 0, 0);
		delta_viscosity[i] = make_float3(0, 0, 0);

		// for each block 
		for (int ii = 0; ii < 27; ii++) {
			int3 blockIdx_nei = blockIdx_i + block_offset[ii];
			if (BlockIdxIsValid(blockIdx_nei, blockDim_i)) {
				int bid_nei = GetBlockIdx1D(blockIdx_nei, blockDim_i);
				// find neighbour particle[j]
				for (int j = block_pidx[bid_nei]; j < block_pidx[bid_nei] + block_pnum[bid_nei]; j++) {
					if (i == j) continue;
					float3 vec_ij = cur_pos[i] - cur_pos[j];
					float len_ij = Norm2(vec_ij);
					len_ij = fmaxf(len_ij, M_EPS);

					//float pol_ker = Poly6Kernel(para->dim, len_ij, para->h, para->poly6_factor);
					//float spi_ker = SpikyGradientKernel(para->dim, len_ij, para->h, para->spiky_grad_factor);
					//float cub_ker = CubicSplineKernel(para->dim, len_ij, para->h, para->cubic_factor3D);
					float cub_ker_deri = CubicSplineKernelDerivative(para->dim, len_ij, para->h, para->cubic_factor3D);

					// Density
					delta_density[i] += para->mass * cub_ker_deri * dot((velocity[i] - velocity[j]), (vec_ij / len_ij));

					// Pressure
					delta_pressure[i] -= para->mass * cub_ker_deri * (vec_ij / len_ij) *
						(pressure[i] / fmaxf(M_EPS, pow(density[i], 2)) + pressure[j] / fmaxf(M_EPS, pow(density[j], 2)));

					// Viscosity
					float v_ij = dot(velocity[i] - velocity[j], vec_ij);
					if (v_ij < 0) {
						float v = -2.0 * para->alpha * para->particle_radius * para->C0 / fmaxf(M_EPS, density[i] + density[j]);
						delta_viscosity[i] -= para->mass * cub_ker_deri * (vec_ij / len_ij) *
							v_ij * v / fmaxf(M_EPS, pow(len_ij, 2) + 0.01 * pow(para->particle_radius, 2));
					}
				}
			}
		}
		threadIdx_i += para->block_thread_num;
	}
}

////////////////////////////////////////////////////////////////////////////////
// 
// Compute delta_velocity and velocity using delta_pressure and delta_viscosity for each particle
// 
////////////////////////////////////////////////////////////////////////////////
__global__ void ComputeVelocity(	WCSPHSystem* para,
									int* block_pidx, int* block_pnum,
									float* density,
									float3* cur_pos, float3* delta_pressure, float3* delta_viscosity, float3* delta_velocity, float3* velocity) {

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = para->block_dim;
	int threadIdx_i = threadIdx.x;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i);
	while (threadIdx_i < block_pnum[bid]) {
		int i = block_pidx[bid] + threadIdx_i; // for each particle[i]
		float3 G = make_float3(0, para->gravity, 0);
		delta_velocity[i] = delta_pressure[i] + delta_viscosity[i] + G;
		velocity[i] += para->time_delta * delta_velocity[i];
		threadIdx_i += para->block_thread_num;
	}
}

////////////////////////////////////////////////////////////////////////////////
//
// Compute new position using velocity for each particle
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ComputePosition(	WCSPHSystem* para,
									int* block_pidx, int* block_pnum,
									float3* cur_pos, float3* next_pos, float3* velocity) {

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = para->block_dim;
	int threadIdx_i = threadIdx.x;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i); 
	while (threadIdx_i < block_pnum[bid]) {
		int i = block_pidx[bid] + threadIdx_i; // for each particle[i]
		next_pos[i] = cur_pos[i] + para->time_delta * velocity[i];
		threadIdx_i += para->block_thread_num;
	}
}


////////////////////////////////////////////////////////////////////////////////
//
// If particle exceed the boundary, confine it to the inside, change the velocity and position
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ConfineToBoundary(	WCSPHSystem* para, hiprandState* devStates,
									int* block_pidx, int* block_pnum, 
									float3* cur_pos, float3* next_pos, float3* velocity) {

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = para->block_dim;
	int threadIdx_i = threadIdx.x;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i);
	while (threadIdx_i < block_pnum[bid]) {
		int i = block_pidx[bid] + threadIdx_i; // for each particle[i]
		// change position if outside
		float3 bmin = make_float3(para->particle_radius);
		float3 bmax = para->box_size - para->particle_radius;

#ifdef CONFINE_RANDOM
		if (next_pos[i].x <= bmin.x) {
			next_pos[i].x = bmin.x + M_EPS * cudaRandomFloat(devStates, i);
		}
		else if (next_pos[i].x >= bmax.x) {
			next_pos[i].x = bmax.x - M_EPS * cudaRandomFloat(devStates, i);
		}

		if (next_pos[i].y <= bmin.y) {
			next_pos[i].y = bmin.y + M_EPS * cudaRandomFloat(devStates, i);
		}
		else if (next_pos[i].y >= bmax.y) {
			next_pos[i].y = bmax.y - M_EPS * cudaRandomFloat(devStates, i);
		}

		if (next_pos[i].z <= bmin.z) {
			next_pos[i].z = bmin.z + M_EPS * cudaRandomFloat(devStates, i);
		}
		else if (next_pos[i].z >= bmax.z) {
			next_pos[i].z = bmax.z - M_EPS * cudaRandomFloat(devStates, i);
		}
		// change velocity
		velocity[i] = (next_pos[i] - cur_pos[i]) / para->time_delta;
#else
		float ETA = para->eta;
		if (next_pos[i].x <= bmin.x) {
			next_pos[i].x = min(bmax.x, bmin.x + (bmin.x - next_pos[i].x) * ETA);
			velocity[i].x = -velocity[i].x * ETA;
		}
		else if (next_pos[i].x >= bmax.x) {
			next_pos[i].x = max(bmin.x, bmax.x - (next_pos[i].x - bmax.x) * ETA);
			velocity[i].x = -velocity[i].x * ETA;
		}

		if (next_pos[i].y <= bmin.y) {
			next_pos[i].y = min(bmax.y, bmin.y + (bmin.y - next_pos[i].y) * ETA);
			velocity[i].y = -velocity[i].y * ETA;
		}
		else if (next_pos[i].y >= bmax.y) {
			next_pos[i].y = max(bmin.y, bmax.y - (next_pos[i].y - bmax.y) * ETA);
			velocity[i].y = -velocity[i].y * ETA;
		}

		if (next_pos[i].z <= bmin.z) {
			next_pos[i].z = min(bmax.z, bmin.z + (bmin.z - next_pos[i].z) * ETA);
			velocity[i].z = -velocity[i].z * ETA;
		}
		else if (next_pos[i].z >= bmax.z) {
			next_pos[i].z = max(bmin.z, bmax.z - (next_pos[i].z - bmax.z) * ETA);
			velocity[i].z = -velocity[i].z * ETA;
		}
#endif // CONFINE_RANDOM

		threadIdx_i += para->block_thread_num;
	}
}


////////////////////////////////////////////////////////////////////////////////
//
// Update the new density, pressure, velocity and position for each particle
//
////////////////////////////////////////////////////////////////////////////////
__global__ void UpdateParticles(	WCSPHSystem* para, 
									int* block_pidx, int* block_pnum,
									float* delta_density, float* density, float* pressure, 
									float3* cur_pos, float3* next_pos, float3* velocity) {

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = para->block_dim;
	int threadIdx_i = threadIdx.x;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i);
	while (threadIdx_i < block_pnum[bid]) {
		int i = block_pidx[bid] + threadIdx_i;

		density[i] += para->time_delta * delta_density[i];

		pressure[i] = PressureUpdate(density[i], para->rho0, para->C0, para->gamma);

		velocity[i] = (next_pos[i] - cur_pos[i]) / para->time_delta;
		velocity[i] *= (1.0 - para->f_air); // air resistence

		cur_pos[i] = next_pos[i];

		threadIdx_i += para->block_thread_num;
	}
}


////////////////////////////////////////////////////////////////////////////////
//
// Use for debug, output the variable value on gpu
//
////////////////////////////////////////////////////////////////////////////////
__global__ void DebugOutput(		WCSPHSystem* para,
									int* particle_bid, int* block_pidx, int* block_pnum,
									float* delta_density, float* density, float* pressure,
									float3* cur_pos, float3* next_pos, float3* delta_pressure, float3* delta_viscocity, float3* delta_velocity, float3* velocity) {
								
	for (int i = 0; i < para->block_num; i++) {
		printf("Block #%d:", i);
		printf("     \n\t block ipdx: %d, block pnum: %d\n", block_pidx[i], block_pnum[i]);
		printf("\n");
	}

	for (int i = 0; i < para->particle_num; i++) {
		printf("Particle #%d:", i);
		printf("     \n\t cur_pos (%f, %f, %f), next_pos (%f, %f, %f), particle_bid: %d\n", cur_pos[i].x, cur_pos[i].y, cur_pos[i].z, next_pos[i].x, next_pos[i].y, next_pos[i].z, particle_bid[i]);
		printf("     \n\t delta_density (%f)\n\t delta_pressure (%f, %f, %f)\n\t delta_velocity (%f, %f, %f)\n", delta_density[i], delta_pressure[i].x, delta_pressure[i].y, delta_pressure[i].z, delta_velocity[i].x, delta_velocity[i].y, delta_velocity[i].z);
		printf("     \n\t density (%f)\n\t pressure (%f)\n\t velocity (%f, %f, %f)\n", density[i], pressure[i], velocity[i].x, velocity[i].y, velocity[i].z);
		printf("\n");
	}
}


////////////////////////////////////////////////////////////////////////////////
//
// Smartly choose the time step to calculate
//
////////////////////////////////////////////////////////////////////////////////
__global__ void AdaptiveStep(		WCSPHSystem* para, 
									float* density, 
									float3* delta_velocity, float3* velocity) {

	float max_v = FLT_MIN;
	float max_a = FLT_MIN;
	float max_r = FLT_MIN;
	for (int i = 0; i < para->particle_num; i++) {
		if (Norm2(velocity[i]) > max_v) {
			max_v = Norm2(velocity[i]);
		}

		if (Norm2(delta_velocity[i]) > max_a) {
			max_a = Norm2(delta_velocity[i]);
		}

		if (density[i] > max_r) {
			max_r = density[i];
		}
	}

	float dt_cfl = para->CFL_v * para->h / max_v;
	float dt_f = para->CFL_a * sqrt(para->h / max_a);
	float dt_a = 0.2 * para->h / (para->C0 * pow(sqrt(max_r / para->rho0), para->gamma));

	para->time_delta = fminf(dt_cfl, fminf(dt_f, dt_a));

}


////////////////////////////////////////////////////////////////////////////////
//
// Export particle information to VBO for drawing
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ExportParticleInfo(	WCSPHSystem* para,
									int* block_pidx, int* block_pnum,
									float3* cur_pos, float3* pos_info, float3* color_info) {

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = para->block_dim;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i);
	int i;
	for (i = block_pidx[bid]; i < block_pidx[bid] + block_pnum[bid]; i++) {
		pos_info[i] = cur_pos[i];
		color_info[i] = make_float3(1, 1, 1);
	}
}


////////////////////////////////////////////////////////////////////////////////
//
// Get next frame information
//
////////////////////////////////////////////////////////////////////////////////
void getNextFrame(WCSPHSystem* para, hipGraphicsResource* position_resource, hipGraphicsResource* color_resource) {
	
	dim3 blocks(para->block_dim.x, para->block_dim.y, para->block_dim.z);
	dim3 threads(para->block_thread_num);

	int num = para->particle_num;

	for (int i = 0; i < para->step_each_frame; i++) {

		//DebugOutput <<<1, 1 >>> (sph_device, particle_bid, block_pidx, block_pnum, delta_density, density, pressure, cur_pos, next_pos, delta_pressure, delta_viscosity, delta_velocity, velocity);
		//hipDeviceSynchronize();

		ComputeBid <<<1, 1 >>> (sph_device, particle_bid, cur_pos);
		hipDeviceSynchronize();

#ifdef CUDA_MEMCPY_ASYNC
		hipStream_t stream[kCudaSortArrayCount];
#endif // CUDA_MEMCPY_ASYNC
		for (int k = 1; k < kCudaSortArrayCount; k++) {
#ifdef CUDA_MEMCPY_ASYNC
			checkCudaErrors(hipStreamCreate(&stream[k]));
			checkCudaErrors(hipMemcpyAsync(particle_bid + num * k, particle_bid, num * sizeof(int), hipMemcpyDeviceToDevice, stream[k]));
#else
			checkCudaErrors(hipMemcpy(particle_bid + num * k, particle_bid, num * sizeof(int), hipMemcpyDeviceToDevice));
#endif // CUDA_MEMCPY_ASYNC
		}

#ifdef CUDA_MEMCPY_ASYNC
		for (int k = 1; k < kCudaSortArrayCount; k++) {
			checkCudaErrors(hipStreamSynchronize(stream[k]));
			checkCudaErrors(hipStreamDestroy(stream[k]));
		}
#endif // CUDA_MEMCPY_ASYNC

		SortParticles <<<kCudaSortArrayCount, 1 >>> (sph_device, particle_bid, density, pressure, cur_pos, velocity);
		hipDeviceSynchronize();

		ComputeBlockIdxPnum <<<1, 1 >>> (sph_device, particle_bid, block_pidx, block_pnum);
		hipDeviceSynchronize();

		ComputeDeltaValue <<<blocks, threads >>> (sph_device, block_pidx, block_pnum, delta_density, density, pressure, block_offset, cur_pos, delta_pressure, delta_viscosity, velocity);
		hipDeviceSynchronize();

		ComputeVelocity <<<blocks, threads >>> (sph_device, block_pidx, block_pnum, density, cur_pos, delta_pressure, delta_viscosity, delta_velocity, velocity);
		hipDeviceSynchronize();

		ComputePosition <<<blocks, threads >>> (sph_device, block_pidx, block_pnum, cur_pos, next_pos, velocity);
		hipDeviceSynchronize();

		ConfineToBoundary <<<blocks, threads >>> (sph_device, devStates, block_pidx, block_pnum, cur_pos, next_pos, velocity);
		hipDeviceSynchronize();

		UpdateParticles <<<blocks, threads >>> (sph_device, block_pidx, block_pnum, delta_density, density, pressure, cur_pos, next_pos, velocity);
		hipDeviceSynchronize();
	}

	float3* pos_info;
	float3* color_info;
	checkCudaErrors(hipGraphicsMapResources(1, &position_resource, 0));
	checkCudaErrors(hipGraphicsMapResources(1, &color_resource, 0));
	size_t pbytes, cbytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&pos_info, &pbytes, position_resource));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&color_info, &cbytes, color_resource));

	ExportParticleInfo <<<blocks, 1 >>> (sph_device, block_pidx, block_pnum, cur_pos, pos_info, color_info);
	hipDeviceSynchronize();

	checkCudaErrors(hipGraphicsUnmapResources(1, &position_resource, 0));
	checkCudaErrors(hipGraphicsUnmapResources(1, &color_resource, 0));
}
