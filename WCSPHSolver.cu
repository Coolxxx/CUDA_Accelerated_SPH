#include "hip/hip_runtime.h"
#include "WCSPHSolver.cuh"
#include "utils/handler.h"

#include <thrust/sort.h>
#include <thrust/execution_policy.h>

//#define DEBUG
//#define CONFINE_RANDOM
#define CUDA_MEMCPY_ASYNC
#define CUDA_MEMSET_ASYNC

const int kCudaSortArrayCount = 4;

#if defined(CUDA_MEMCPY_ASYNC) || defined(CUDA_MEMCPY_ASYNC)
const int kCudaMemcpyTime = 7;
#endif // defined(CUDA_MEMCPY_ASYNC) || defined(CUDA_MEMCPY_ASYNC)


////////////////////////////////////////////////////////////////////////////////
// Device array declare
////////////////////////////////////////////////////////////////////////////////

WCSPHSystem* sph_device = NULL;

int* particle_bid = NULL; // each particle belongs to which block
int* block_pidx = NULL; // first particle index in grid
int* block_pnum = NULL; // particle number in grid

hiprandState* devStates = NULL;

float3* color = NULL; // color of particles
float3* cur_pos = NULL;
float3* next_pos = NULL;

float* density = NULL;
float* delta_density = NULL;

float* pressure = NULL;
float3* delta_pressure = NULL;

float3* delta_viscosity = NULL;

float* velo_min = NULL;
float* velo_max = NULL;
float* velocity_len = NULL;
float3* velocity = NULL;
float3* delta_velocity = NULL;


////////////////////////////////////////////////////////////////////////////////
//
// Init CUDA Device System
//
////////////////////////////////////////////////////////////////////////////////
void InitDeviceSystem(WCSPHSystem* para, float* dens_init, float3* pos_init, float3* velo_init) {

#ifdef DEBUG
	std::cout << "Do InitDeviceSystem" << std::endl;
#endif // DEBUG

	int num = para->particle_num;
#if defined (CUDA_MEMCPY_ASYNC) || defined (CUDA_MEMSET_ASYNC)
	hipStream_t stream[kCudaMemcpyTime];
	int streamnum = 0;
	for (int i = 0; i < kCudaMemcpyTime; i++) {
		checkCudaErrors(hipStreamCreate(&stream[i]));
	}
#endif // CUDA_MEMCPY_ASYNC || CUDA_MEMSET_ASYNC

	checkCudaErrors(hipMalloc((void**)&sph_device, sizeof(WCSPHSystem)));
#ifdef CUDA_MEMCPY_ASYNC
	checkCudaErrors(hipMemcpyAsync(sph_device, para, sizeof(WCSPHSystem), hipMemcpyHostToDevice, stream[streamnum++]));
#else
	checkCudaErrors(hipMemcpy(sph_device, para, sizeof(WCSPHSystem), hipMemcpyHostToDevice));
#endif // CUDA_MEMCPY_ASYNC
	

	checkCudaErrors(hipMalloc((void**)&particle_bid, kCudaSortArrayCount * num * sizeof(int)));

	checkCudaErrors(hipMalloc((void**)&block_pidx, para->grid_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&block_pnum, para->grid_size * sizeof(int)));


	checkCudaErrors(hipMalloc((void**)&devStates, num * sizeof(hiprandState)));

	checkCudaErrors(hipMalloc((void**)&color, num * sizeof(float3)));


	checkCudaErrors(hipMalloc((void**)&cur_pos, num * sizeof(float3)));
#ifdef CUDA_MEMCPY_ASYNC
	checkCudaErrors(hipMemcpyAsync(cur_pos, pos_init, num * sizeof(float3), hipMemcpyHostToDevice, stream[streamnum++]));
#else
	checkCudaErrors(hipMemcpy(cur_pos, pos_init, num * sizeof(float3), hipMemcpyHostToDevice));
#endif // CUDA_MEMCPY_ASYNC


	checkCudaErrors(hipMalloc((void**)&next_pos, num * sizeof(float3)));
#ifdef CUDA_MEMSET_ASYNC
	checkCudaErrors(hipMemsetAsync(next_pos, 0, num * sizeof(float3), stream[3]));
#else
	checkCudaErrors(hipMemset(next_pos, 0, num * sizeof(float3)));
#endif // CUDA_MEMSET_ASYNC

	checkCudaErrors(hipMalloc((void**)&density, num * sizeof(float)));
#ifdef CUDA_MEMCPY_ASYNC
	checkCudaErrors(hipMemcpyAsync(density, dens_init, num * sizeof(float), hipMemcpyHostToDevice, stream[streamnum++]));
#else
	checkCudaErrors(hipMemcpy(density, dens_init, num * sizeof(float), hipMemcpyHostToDevice));
#endif // CUDA_MEMCPY_ASYNC

	checkCudaErrors(hipMalloc((void**)&delta_density, num * sizeof(float)));

	checkCudaErrors(hipMalloc((void**)&pressure, num * sizeof(float)));
#ifdef CUDA_MEMSET_ASYNC
	checkCudaErrors(hipMemsetAsync(pressure, 0, num * sizeof(float), stream[5]));
#else
	checkCudaErrors(hipMemset(pressure, 0, num * sizeof(float)));
#endif // CUDA_MEMSET_ASYNC

	checkCudaErrors(hipMalloc((void**)&delta_pressure, num * sizeof(float3)));

	checkCudaErrors(hipMalloc((void**)&delta_viscosity, num * sizeof(float3)));
	

	checkCudaErrors(hipMalloc((void**)&velo_min, sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&velo_max, sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&velocity_len, num * sizeof(float)));

	checkCudaErrors(hipMalloc((void**)&velocity, num * sizeof(float3)));
#ifdef CUDA_MEMCPY_ASYNC
	checkCudaErrors(hipMemcpyAsync(velocity, velo_init, num * sizeof(float3), hipMemcpyHostToDevice, stream[streamnum++]));
#else
	checkCudaErrors(hipMemcpy(velocity, velo_init, num * sizeof(float3), hipMemcpyHostToDevice));
#endif // CUDA_MEMCPY_ASYNC

	checkCudaErrors(hipMalloc((void**)&delta_velocity, num * sizeof(float3)));

	

#if defined (CUDA_MEMCPY_ASYNC) || defined (CUDA_MEMSET_ASYNC)
	for (int i = 0; i < kCudaMemcpyTime; i++) {
		checkCudaErrors(hipStreamSynchronize(stream[i]));
		checkCudaErrors(hipStreamDestroy(stream[i]));
	}
#endif // CUDA_MEMCPY_ASYNC || CUDA_MEMSET_ASYNC

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

#ifdef DEBUG
	std::cout << "Finish InitDeviceSystem" << std::endl;
#endif // DEBUG
}

////////////////////////////////////////////////////////////////////////////////
//
// Free CUDA Device System
//
////////////////////////////////////////////////////////////////////////////////
void FreeDeviceSystem(WCSPHSystem* para) {

#ifdef DEBUG
	std::cout << "Do FreeDeviceSystem" << std::endl;
#endif // DEBUG
	delete para;

	checkCudaErrors(hipFree(sph_device));

	checkCudaErrors(hipFree(particle_bid));
	checkCudaErrors(hipFree(block_pidx));
	checkCudaErrors(hipFree(block_pnum));

	checkCudaErrors(hipFree(devStates));

	checkCudaErrors(hipFree(color));

	checkCudaErrors(hipFree(cur_pos));
	checkCudaErrors(hipFree(next_pos));

	checkCudaErrors(hipFree(density));
	checkCudaErrors(hipFree(delta_density));

	checkCudaErrors(hipFree(pressure));
	checkCudaErrors(hipFree(delta_pressure));

	checkCudaErrors(hipFree(delta_viscosity));

	checkCudaErrors(hipFree(velo_min));
	checkCudaErrors(hipFree(velo_max));
	checkCudaErrors(hipFree(velocity_len));
	checkCudaErrors(hipFree(velocity));
	checkCudaErrors(hipFree(delta_velocity));

#ifdef DEBUG
	std::cout << "Finish InitDeviceSystem" << std::endl;
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
// CUDA function are implemented here
////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
//
// Compute which block each particle belongs to
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ComputeBid(			WCSPHSystem* para,
									int* particle_bid,
									float3* cur_pos) {

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Do ComputeBid\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	// compute block_id for each particle
	int i = GetBlockIdx1D(blockIdx, gridDim) * GetDimTotalSize(blockDim) + threadIdx.x;
	while (i < para->particle_num) {
		// compute particle position inside which bidx block
		int3 bidx = make_int3(cur_pos[i] / para->block_length);
		particle_bid[i] = GetBlockIdx1D(bidx, para->grid_dim);
		i += GetDimTotalSize(gridDim) * GetDimTotalSize(blockDim); // gridSize * blockSize
	}



#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Finish ComputeBid\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
//
// Use Radix sort to place particle in block order
//
////////////////////////////////////////////////////////////////////////////////
__global__ void SortParticles(		WCSPHSystem* para,
									int* particle_bid,
									float* density, float* pressure,
									float3* cur_pos, float3* velocity) {

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Do SortParticles\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	int num = para->particle_num;
	if (blockIdx.x == 0) {
		thrust::stable_sort_by_key(thrust::device, particle_bid + num * 0, particle_bid + num * 1, cur_pos);
	}
	else if (blockIdx.x == 1) {
		thrust::stable_sort_by_key(thrust::device, particle_bid + num * 1, particle_bid + num * 2, density);
	}
	else if (blockIdx.x == 2) {
		thrust::stable_sort_by_key(thrust::device, particle_bid + num * 2, particle_bid + num * 3, pressure);
	}
	else if (blockIdx.x == 3) {
		thrust::stable_sort_by_key(thrust::device, particle_bid + num * 3, particle_bid + num * 4, velocity);
	}
	//else if (blockIdx.x == 4) {
	//	thrust::stable_sort_by_key(thrust::device, particle_bid + num * 4, particle_bid + num * 5, next_pos);
	//}

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Finish SortParticles\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
//
// Compute the index of first particle and the total particle number in each block 
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ComputeBlockIdxPnum(WCSPHSystem* para,
									int* particle_bid, int* block_pidx, int* block_pnum) {

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Do ComputeBlockIdxPnum\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	for (int i = 0; i < para->grid_size; i++) {
		block_pidx[i] = -1;
		block_pnum[i] = 0;
	}

	for (int i = 0; i < para->particle_num; i++) {
		if (i == 0 || particle_bid[i] != particle_bid[i - 1]) {
			block_pidx[particle_bid[i]] = i;
		}
		block_pnum[particle_bid[i]]++;
		//if (block_pnum[particle_bid[i]] > para->block_size)
		//	printf("Block %d ERROR, exceed threads number\n", particle_bid[i]);
	}

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Finish ComputeBlockIdxPnum\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
//
// Compute delta value of density, pressure and viscosity for each particle
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ComputeDeltaValue(	WCSPHSystem* para,
									int* block_pidx, int* block_pnum,
									float* delta_density, float* density,  float* pressure,
									float3* cur_pos, float3* delta_pressure, float3* delta_viscosity, float3* velocity) {

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Do ComputeDeltaValue\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = make_int3(para->grid_dim);
	int threadIdx_i = threadIdx.x;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i);

	while (threadIdx_i < block_pnum[bid]) {
		// for each particle[i]
		int i = block_pidx[bid] + threadIdx_i;
		// Initialize
		delta_density[i] = 0.0;
		delta_pressure[i] = make_float3(0, 0, 0);
		delta_viscosity[i] = make_float3(0, 0, 0);

		// for each block 
		for (int ii = 0; ii < 27; ii++) {
			int3 blockIdx_nei = blockIdx_i + make_int3(ii / 9 - 1, (ii % 9) / 3 - 1, ii % 3 - 1);
			if (BlockIdxIsValid(blockIdx_nei, blockDim_i)) {
				int bid_nei = GetBlockIdx1D(blockIdx_nei, blockDim_i);
				// find neighbour particle[j]
#pragma unroll
				for (int j = block_pidx[bid_nei]; j < block_pidx[bid_nei] + block_pnum[bid_nei]; j++) {
					if (i == j) continue;
					float3 vec_ij = cur_pos[i] - cur_pos[j];
					float len_ij = Norm2(vec_ij);
					len_ij = fmaxf(len_ij, M_EPS);

					//float pol_ker = Poly6Kernel(para->dim, len_ij, para->h, para->poly6_factor);
					//float spi_ker = SpikyGradientKernel(para->dim, len_ij, para->h, para->spiky_grad_factor);
					float cub_ker = CubicSplineKernel(para->dim, len_ij, para->h, para->cubic_factor3D);
					float cub_ker_deri = CubicSplineKernelDerivative(para->dim, len_ij, para->h, para->cubic_factor3D);

					// Density (Continuity equation, summation approach)
					delta_density[i] += para->mass * cub_ker;

					//// Density (Continuity equation, differential update)
					//delta_density[i] += para->mass * cub_ker_deri * dot((velocity[i] - velocity[j]), (vec_ij / len_ij));

					// Pressure (Momentum equation)
					delta_pressure[i] -= para->mass * cub_ker_deri * (vec_ij / len_ij) *
						(pressure[i] / fmaxf(M_EPS, pow(density[i], 2)) + pressure[j] / fmaxf(M_EPS, pow(density[j], 2)));

					// Viscosity
					float v_ij = dot(velocity[i] - velocity[j], vec_ij);
					if (v_ij < 0) {
						float viscous = -2.0 * para->alpha * para->h * para->C_s / fmaxf(M_EPS, density[i] + density[j]);
						delta_viscosity[i] -= para->mass * cub_ker_deri * (vec_ij / len_ij) * 
							viscous * v_ij / fmaxf(M_EPS, pow(len_ij, 2) + 0.01 * pow(para->h, 2));
					}
				}
			}
		}
		threadIdx_i += para->block_size;
	}

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Finish ComputeDeltaValue\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
// 
// Compute delta_velocity and velocity using delta_pressure and delta_viscosity for each particle
// 
////////////////////////////////////////////////////////////////////////////////
__global__ void ComputeVelocity(	WCSPHSystem* para,
									int* block_pidx, int* block_pnum,
									float* density,
									float3* cur_pos, float3* delta_pressure, float3* delta_viscosity, float3* delta_velocity, float3* velocity) {

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Do ComputeVelocity\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = make_int3(para->grid_dim);
	int threadIdx_i = threadIdx.x;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i);
	while (threadIdx_i < block_pnum[bid]) {
		int i = block_pidx[bid] + threadIdx_i; // for each particle[i]
		float3 G = make_float3(0, para->gravity, 0);
		// velocity (Momentum equation)
		delta_velocity[i] = delta_pressure[i] + delta_viscosity[i] + G;
		velocity[i] += para->time_delta * delta_velocity[i];
		threadIdx_i += para->block_size;
	}

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Finish ComputeVelocity\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
//
// Compute new position using velocity for each particle
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ComputePosition(	WCSPHSystem* para,
									int* block_pidx, int* block_pnum,
									float3* cur_pos, float3* next_pos, float3* velocity) {

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Do ComputePosition\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = make_int3(para->grid_dim);
	int threadIdx_i = threadIdx.x;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i); 
	while (threadIdx_i < block_pnum[bid]) {
		int i = block_pidx[bid] + threadIdx_i; // for each particle[i]
		next_pos[i] = cur_pos[i] + para->time_delta * velocity[i];
		threadIdx_i += para->block_size;
	}

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Finish ComputePosition\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
//
// If particle exceed the boundary, confine it to the inside, change the velocity and position
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ConfineToBoundary(	WCSPHSystem* para, hiprandState* devStates,
									int* block_pidx, int* block_pnum, 
									float3* cur_pos, float3* next_pos, float3* velocity) {

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = make_int3(para->grid_dim);
	int threadIdx_i = threadIdx.x;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i);
	while (threadIdx_i < block_pnum[bid]) {
		int i = block_pidx[bid] + threadIdx_i; // for each particle[i]
		// change position if outside
		float3 bmin = make_float3(para->particle_radius);
		float3 bmax = para->box_length - para->particle_radius;

#ifdef CONFINE_RANDOM
		if (next_pos[i].x <= bmin.x) {
			next_pos[i].x = bmin.x + M_EPS * cudaRandomFloat(devStates, i);
		}
		else if (next_pos[i].x >= bmax.x) {
			next_pos[i].x = bmax.x - M_EPS * cudaRandomFloat(devStates, i);
		}

		if (next_pos[i].y <= bmin.y) {
			next_pos[i].y = bmin.y + M_EPS * cudaRandomFloat(devStates, i);
		}
		else if (next_pos[i].y >= bmax.y) {
			next_pos[i].y = bmax.y - M_EPS * cudaRandomFloat(devStates, i);
		}

		if (next_pos[i].z <= bmin.z) {
			next_pos[i].z = bmin.z + M_EPS * cudaRandomFloat(devStates, i);
		}
		else if (next_pos[i].z >= bmax.z) {
			next_pos[i].z = bmax.z - M_EPS * cudaRandomFloat(devStates, i);
		}
		// change velocity
		velocity[i] = (next_pos[i] - cur_pos[i]) / para->time_delta;
#else
		float ETA = para->eta;
		if (next_pos[i].x <= bmin.x) {
			next_pos[i].x = min(bmax.x, bmin.x + (bmin.x - next_pos[i].x) * ETA);
			velocity[i].x = -velocity[i].x * ETA;
		}
		else if (next_pos[i].x >= bmax.x) {
			next_pos[i].x = max(bmin.x, bmax.x - (next_pos[i].x - bmax.x) * ETA);
			velocity[i].x = -velocity[i].x * ETA;
		}

		if (next_pos[i].y <= bmin.y) {
			next_pos[i].y = min(bmax.y, bmin.y + (bmin.y - next_pos[i].y) * ETA);
			velocity[i].y = -velocity[i].y * ETA;
		}
		else if (next_pos[i].y >= bmax.y) {
			next_pos[i].y = max(bmin.y, bmax.y - (next_pos[i].y - bmax.y) * ETA);
			velocity[i].y = -velocity[i].y * ETA;
		}

		if (next_pos[i].z <= bmin.z) {
			next_pos[i].z = min(bmax.z, bmin.z + (bmin.z - next_pos[i].z) * ETA);
			velocity[i].z = -velocity[i].z * ETA;
		}
		else if (next_pos[i].z >= bmax.z) {
			next_pos[i].z = max(bmin.z, bmax.z - (next_pos[i].z - bmax.z) * ETA);
			velocity[i].z = -velocity[i].z * ETA;
		}
#endif // CONFINE_RANDOM

		threadIdx_i += para->block_size;
	}
}


////////////////////////////////////////////////////////////////////////////////
//
// Update the new density, pressure, velocity and position for each particle
//
////////////////////////////////////////////////////////////////////////////////
__global__ void UpdateParticles(	WCSPHSystem* para,
									int* block_pidx, int* block_pnum,
									float* delta_density, float* density, float* pressure, float* velocity_len,
									float3* cur_pos, float3* next_pos, float3* velocity) {

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Do UpdateParticles\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = make_int3(para->grid_dim);
	int threadIdx_i = threadIdx.x;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i);
	while (threadIdx_i < block_pnum[bid]) {
		int i = block_pidx[bid] + threadIdx_i;

		density[i] += para->time_delta * delta_density[i];

		pressure[i] = PressureUpdate(density[i], para->rho_0, para->C_s, para->gamma);

#ifdef CONFINE_RANDOM
		velocity[i] = (next_pos[i] - cur_pos[i]) / para->time_delta;
#endif // CONFINE_RANDOM

		velocity[i] *= (1.0 - para->f_air); // air resistence

		velocity_len[i] = Norm2(velocity[i]);

		cur_pos[i] = next_pos[i];

		threadIdx_i += para->block_size;
	}

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Finish UpdateParticles\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

}


////////////////////////////////////////////////////////////////////////////////
//
// Use for debug, output the variable value on gpu
//
////////////////////////////////////////////////////////////////////////////////
__global__ void DebugOutput(		WCSPHSystem* para,
									int* particle_bid, int* block_pidx, int* block_pnum,
									float* delta_density, float* density, float* pressure,
									float3* cur_pos, float3* next_pos, float3* delta_pressure, float3* delta_viscocity, float3* delta_velocity, float3* velocity) {
								
	//for (int i = 0; i < para->grid_size; i++) {
	//	printf("Block #%d:", i);
	//	printf("     \n\t block ipdx: %d, block pnum: %d\n", block_pidx[i], block_pnum[i]);
	//	printf("\n");
	//}

	for (int i = 0; i < para->particle_num; i++) {
		printf("Particle #%d:", i);
		printf("\n\t particle_bid: %d\n\t cur_pos (%f, %f, %f)\n\t next_pos (%f, %f, %f)\n", particle_bid[i], cur_pos[i].x, cur_pos[i].y, cur_pos[i].z, next_pos[i].x, next_pos[i].y, next_pos[i].z);
		printf("\n\t delta_density (%f)\n\t delta_pressure (%f, %f, %f)\n\t delta_viscosity (%f, %f, %f)\n\t delta_velocity (%f, %f, %f)\n", delta_density[i], delta_pressure[i].x, delta_pressure[i].y, delta_pressure[i].z, delta_viscocity[i].x, delta_viscocity[i].y, delta_viscocity[i].z, delta_velocity[i].x, delta_velocity[i].y, delta_velocity[i].z);
		printf("\n\t density (%f)\n\t pressure (%f)\n\t velocity (%f, %f, %f)\n", density[i], pressure[i], velocity[i].x, velocity[i].y, velocity[i].z);
		printf("\n");
	}
}


////////////////////////////////////////////////////////////////////////////////
//
// Smartly choose the time step to calculate
//
////////////////////////////////////////////////////////////////////////////////
__global__ void AdaptiveStep(		WCSPHSystem* para, 
									float* density, 
									float3* delta_velocity, float3* velocity) {

	float max_v = FLT_MIN;
	float max_a = FLT_MIN;
	float max_r = FLT_MIN;
	for (int i = 0; i < para->particle_num; i++) {
		if (Norm2(velocity[i]) > max_v) {
			max_v = Norm2(velocity[i]);
		}

		if (Norm2(delta_velocity[i]) > max_a) {
			max_a = Norm2(delta_velocity[i]);
		}

		if (density[i] > max_r) {
			max_r = density[i];
		}
	}

	float dt_cfl = para->CFL_v * para->h / max_v;
	float dt_f = para->CFL_a * sqrt(para->h / max_a);
	float dt_a = 0.2 * para->h / (para->C_s * pow(sqrt(max_r / para->rho_0), para->gamma));

	para->time_delta = fminf(dt_cfl, fminf(dt_f, dt_a));

}


////////////////////////////////////////////////////////////////////////////////
//
// Find maximum and minimum value of velocity_len for each particle
//
////////////////////////////////////////////////////////////////////////////////
__global__ void FindVelocityLenMinMax(unsigned int blockSize, float* velocity_len, float* g_odata, unsigned int num, bool findmin) {

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Do FindVelocityLenMinMax\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	if (findmin)
		sdata[tid] = 1e20;
	else sdata[tid] = 0;
	pfunc func = find_minmax[findmin];

	while (i < num) {
		sdata[tid] = func(sdata[tid], velocity_len[i]);
		if (i + blockSize < num)
			sdata[tid] = func(sdata[tid], velocity_len[i + blockSize]);
		i += gridSize;
	}
	__syncthreads();
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] = func(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] = func(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) { sdata[tid] = func(sdata[tid], sdata[tid +  64]); } __syncthreads(); }
	if (tid < 32) { FindMinMaxWarpReduce(blockSize, sdata, tid, func); }
	if (tid == 0) { g_odata[blockIdx.x] = sdata[0]; }
	if (tid == 0) { printf("velocity_max: %f\n", g_odata[blockIdx.x]); }

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Finish FindVelocityLenMinMax\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG
}


////////////////////////////////////////////////////////////////////////////////
//
// Export particle information to VBO for drawing, blue(0, 0, 1) is slow, white(1, 1, 1) is fast
//
////////////////////////////////////////////////////////////////////////////////
__global__ void ExportParticleInfo(	WCSPHSystem* para,
									int* block_pidx, int* block_pnum,
									float* velocity_len, float* velo_min, float* velo_max,
									float3* cur_pos, float3* pos_info, float3* color_info) {

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Do ExportParticleInfo\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

	int3 blockIdx_i = make_int3(blockIdx.x, blockIdx.y, blockIdx.z);
	int3 blockDim_i = make_int3(para->grid_dim);
	int threadIdx_i = threadIdx.x;
	int bid = GetBlockIdx1D(blockIdx_i, blockDim_i);
	while (threadIdx_i < block_pnum[bid]) {
		int i = block_pidx[bid] + threadIdx_i;
		pos_info[i] = cur_pos[i];
		float percent = NormalizeTo01(velocity_len[i], para->velo_draw_min, para->velo_draw_max);
		//float percent = NormalizeTo01(velocity_len[i], *velo_min, *velo_max);
		color_info[i] = make_float3(percent, percent, 1.0);
		threadIdx_i += para->block_size;
	}

#ifdef DEBUG
	if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0)
		printf("Block #(%d,%d,%d) Finish ExportParticleInfo\n", blockIdx.x, blockIdx.y, blockIdx.z);
#endif // DEBUG

}


////////////////////////////////////////////////////////////////////////////////
//
// Get next frame information
//
////////////////////////////////////////////////////////////////////////////////
void getNextFrame(WCSPHSystem* para, hipGraphicsResource* position_resource, hipGraphicsResource* color_resource) {
	
	dim3 blocks(para->grid_dim.x, para->grid_dim.y, para->grid_dim.z);
	dim3 threads(para->block_size);

	unsigned int num = para->particle_num;
	unsigned int thread_num = para->block_size;

	for (int i = 0; i < para->step_each_frame; i++) {

		//DebugOutput <<<1, 1 >>> (sph_device, particle_bid, block_pidx, block_pnum, delta_density, density, pressure, cur_pos, next_pos, delta_pressure, delta_viscosity, delta_velocity, velocity);
		//hipDeviceSynchronize();

		//ComputeBid <<<1, 1 >>> (sph_device, particle_bid, cur_pos);
		ComputeBid <<<blocks, threads >>> (sph_device, particle_bid, cur_pos);
		hipDeviceSynchronize();

#ifdef CUDA_MEMCPY_ASYNC
		hipStream_t stream[kCudaSortArrayCount];
#endif // CUDA_MEMCPY_ASYNC
		for (int k = 1; k < kCudaSortArrayCount; k++) {
#ifdef CUDA_MEMCPY_ASYNC
			checkCudaErrors(hipStreamCreate(&stream[k]));
			checkCudaErrors(hipMemcpyAsync(particle_bid + num * k, particle_bid, num * sizeof(int), hipMemcpyDeviceToDevice, stream[k]));
#else
			checkCudaErrors(hipMemcpy(particle_bid + num * k, particle_bid, num * sizeof(int), hipMemcpyDeviceToDevice));
#endif // CUDA_MEMCPY_ASYNC
		}

#ifdef CUDA_MEMCPY_ASYNC
		for (int k = 1; k < kCudaSortArrayCount; k++) {
			checkCudaErrors(hipStreamSynchronize(stream[k]));
			checkCudaErrors(hipStreamDestroy(stream[k]));
		}
#endif // CUDA_MEMCPY_ASYNC

		SortParticles <<<kCudaSortArrayCount, 1 >>> (sph_device, particle_bid, density, pressure, cur_pos, velocity);
		hipDeviceSynchronize();

		ComputeBlockIdxPnum <<<1, 1 >>> (sph_device, particle_bid, block_pidx, block_pnum);
		hipDeviceSynchronize();

		ComputeDeltaValue <<<blocks, threads >>> (sph_device, block_pidx, block_pnum, delta_density, density, pressure, cur_pos, delta_pressure, delta_viscosity, velocity);
		hipDeviceSynchronize();

		ComputeVelocity <<<blocks, threads >>> (sph_device, block_pidx, block_pnum, density, cur_pos, delta_pressure, delta_viscosity, delta_velocity, velocity);
		hipDeviceSynchronize();

		ComputePosition <<<blocks, threads >>> (sph_device, block_pidx, block_pnum, cur_pos, next_pos, velocity);
		hipDeviceSynchronize();

		ConfineToBoundary <<<blocks, threads >>> (sph_device, devStates, block_pidx, block_pnum, cur_pos, next_pos, velocity);
		hipDeviceSynchronize();

		UpdateParticles <<<blocks, threads >>> (sph_device, block_pidx, block_pnum, delta_density, density, pressure, velocity_len, cur_pos, next_pos, velocity);
		hipDeviceSynchronize();
	}

	//FindVelocityLenMinMax <<<1, threads, thread_num * sizeof(float)  >>> (thread_num, velocity_len, velo_min, num, true); // find min
	//hipDeviceSynchronize();

	//FindVelocityLenMinMax <<<1, threads, thread_num * sizeof(float)  >>> (thread_num, velocity_len, velo_max, num, false); // find max
	//hipDeviceSynchronize();

	float3* pos_info;
	float3* color_info;
	checkCudaErrors(hipGraphicsMapResources(1, &position_resource));
	checkCudaErrors(hipGraphicsMapResources(1, &color_resource));
	hipDeviceSynchronize();
	size_t pbytes, cbytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&pos_info, &pbytes, position_resource));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&color_info, &cbytes, color_resource));
	hipDeviceSynchronize();
	
	ExportParticleInfo <<<blocks, threads >>> (sph_device, block_pidx, block_pnum, velocity_len, velo_min, velo_max, cur_pos, pos_info, color_info);
	hipDeviceSynchronize();
	
	checkCudaErrors(hipGraphicsUnmapResources(1, &position_resource));
	checkCudaErrors(hipGraphicsUnmapResources(1, &color_resource));
	hipDeviceSynchronize();
}
